
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>  
#include <iostream>

#define SIZE 1024
__global__ void scanKernel(int* d_out, int* d_in) {
    __shared__ int temp[SIZE]; 

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    temp[tid] = d_in[tid];
    __syncthreads();


    int value = 0;
    for (int i = 0; i <= tid; i++) {
        value += temp[i];
    }

    d_out[tid] = value;
}

int main() {
    int* h_input = (int*)malloc(sizeof(int) * SIZE);
    int* h_output = (int*)malloc(sizeof(int) * SIZE);
    int* d_input;
    int* d_output;

    
    for (int i = 0; i < SIZE; i++) {
        h_input[i] = 1;  
    }
    

   
    hipMalloc((void**)&d_input, sizeof(int) * SIZE);
    hipMalloc((void**)&d_output, sizeof(int) * SIZE);

 
    hipMemcpy(d_input, h_input, sizeof(int) * SIZE, hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();
    scanKernel<<<1, SIZE>>>(d_output, d_input);
    auto end = std::chrono::high_resolution_clock::now();
    hipMemcpy(h_output, d_output, sizeof(int) * SIZE, hipMemcpyDeviceToHost);
    
    std::chrono::duration<double> elapsed = end - start;
    printf("Scan result:\n");
    for (int i = 0; i < SIZE; i++) {
        printf("%d ", h_output[i]);
    }
    printf("\n");

    std::cout << "Time taken for naive GPU prefix sum: " << elapsed.count() << " seconds" << std::endl;
    hipFree(d_input);
    hipFree(d_output);


    free(h_input);
    free(h_output);

    return 0;
}
