#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>  

__global__ void recursiveDoublingPrefixSum(int* input, int* output, int n) {
    extern __shared__ int temp[];  
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    temp[threadIdx.x] = input[tid];

    __syncthreads();
    
    for (int offset = 1; offset < blockDim.x; offset *= 2) {
        int tempVal = 0;
        if (threadIdx.x >= offset) {
            tempVal = temp[threadIdx.x - offset];
        }
        __syncthreads();
        temp[threadIdx.x] += tempVal;
        __syncthreads();
    }

    // Store the result
    output[tid] = temp[threadIdx.x];
}

int main() {
    int SIZE = 10240;  // Example size
    int* h_input = (int*)malloc(sizeof(int) * SIZE);
    int* h_output = (int*)malloc(sizeof(int) * SIZE);
    int* d_input;
    int* d_output;

    for (int i = 0; i < SIZE; i++) {
        h_input[i] = 1;  
    }
   

    hipMalloc((void**)&d_input, SIZE * sizeof(int));
    hipMalloc((void**)&d_output, SIZE * sizeof(int));
    hipMemcpy(d_input, h_input, SIZE * sizeof(int), hipMemcpyHostToDevice);
    auto start = std::chrono::high_resolution_clock::now();
    recursiveDoublingPrefixSum<<<1, SIZE, SIZE * sizeof(int)>>>(d_input, d_output, SIZE);
   
    auto end = std::chrono::high_resolution_clock::now();
    hipMemcpy(h_output, d_output, SIZE * sizeof(int), hipMemcpyDeviceToHost);
    printf("%s\n", hipGetErrorString(hipGetLastError()));

    
    
    std::chrono::duration<double> elapsed = end - start;

    // printf("Prefix Sum Output:\n");
    // for (int i = 0; i < SIZE; i++) {
    //     printf("%d ", h_output[i]);
    // }
    // printf("\n");
    std::cout << "Time taken for naive GPU prefix sum: " << elapsed.count() << " seconds" << std::endl;
    hipFree(d_input);
    hipFree(d_output);
    return 0;
}
